#include "hip/hip_runtime.h"
﻿
// some taken from OptiX_Utility
// https://github.com/shocker-0x15/OptiX_Utility/blob/master/LICENSE.md
// and from Shocker GfxExp
// https://github.com/shocker-0x15/GfxEx

#include "principledDisney_ripr.h"




RT_PIPELINE_LAUNCH_PARAMETERS ripr_shared::PipelineLaunchParameters ripr_plp;

// R2 Sequence Sampling Implementation
// ===================================
// 
// This implementation uses the R2 sequence for pixel sampling instead of random jittering.
// The R2 sequence provides optimal 2D coverage with minimal discrepancy.
//
// Benefits over Random Sampling:
// - Better Distribution: Optimal 2D coverage with minimal gaps/clusters
// - Deterministic: Reproducible results, easier debugging
// - Low Discrepancy: Converges faster than random sampling
// - No Clustering: Avoids the clumping artifacts of random sampling
// - Temporal Stability: Smooth progression between frames
//
// The Math:
// The plastic constant g ≈ 1.324717 is the unique real solution to x³ = x + 1.
// This number has special properties that make it ideal for generating well-distributed
// 2D point sets. The R2 sequence is a recent discovery (2018) that provides better
// 2D coverage than traditional sequences like Halton or Sobol.
//
// Implementation:
// - Each pixel gets a unique sample based on its position
// - Samples change every frame using frameNumber offset
// - Sample index = pixelIndex + frameNumber * (width * height)
// - This ensures no correlation between adjacent pixels and temporal variation

// Math constants
#ifndef M_PIf
#define M_PIf 3.14159265358979323846f
#endif

// R2 sequence constants
// g = plastic constant = solution to x^3 = x + 1
// Provides optimal 2D coverage with minimal discrepancy
__constant__ float R2_G = 1.32471795724474602596f;
__constant__ float R2_A1 = 0.7548776662466927f;  // 1/g
__constant__ float R2_A2 = 0.5698402909980532f;  // 1/(g*g)

// Helper function for fractional part
CUDA_DEVICE_FUNCTION CUDA_INLINE float fract(float x) {
    return x - floorf(x);
}

// R2 sequence generator
CUDA_DEVICE_FUNCTION CUDA_INLINE float2 R2Sequence(uint32_t index) {
    return make_float2(
        fract(R2_A1 * index),
        fract(R2_A2 * index)
    );
}

// basic math from https://github.com/jbikker/lighthouse2
// Samples a point on a polygonal lens shape (9-sided polygon for smooth bokeh)
CUDA_DEVICE_FUNCTION Vector3D sampleLensPoint (float r0, float r1, float lensSize)
{
    // Convert first random number to select polygon edge
    const float blade = (int)(r0 * 9);
    float r2 = (r0 - blade * (1.0f / 9.0f)) * 9.0f;

    // Get vertices of the selected edge
    float x1, y1, x2, y2;
    float angle1 = blade * M_PI / 4.5f;
    float angle2 = (blade + 1.0f) * M_PI / 4.5f;
    x1 = cos (angle1);
    y1 = sin (angle1);
    x2 = cos (angle2);
    y2 = sin (angle2);

    // Handle point reflection to ensure uniform sampling
    if ((r1 + r2) > 1)
    {
        r1 = 1.0f - r1;
        r2 = 1.0f - r2;
    }

    // Interpolate between vertices
    float x = x1 * r1 + x2 * r2;
    float y = y1 * r1 + y2 * r2;

    return Vector3D (x * lensSize, y * lensSize, 0.0f);
}

// basic math from https://github.com/jbikker/lighthouse2
// Generate camera ray with configurable focus distance and lens size
CUDA_DEVICE_FUNCTION void generateCameraRay (
    PCG32RNG& rng,
    const PerspectiveCamera& camera,
    const Point2D& pixel,
    Point3D* origin,
    Vector3D* direction)
{
    // Calculate position on image plane
    float h = 2.0f * std::tan (camera.fovY * 0.5f);
    float w = camera.aspect * h;
    Vector3D imagePlanePoint = (w * (0.5f - pixel.x)) * camera.orientation.c0 +
                               (h * (0.5f - pixel.y)) * camera.orientation.c1 +
                               camera.orientation.c2;

    if (camera.lensSize > 0.0f)
    {
        // Generate random point on lens
        float r0 = rng.getFloat0cTo1o();
        float r1 = rng.getFloat0cTo1o();
        Vector3D lensPoint = sampleLensPoint (r0, r1, camera.lensSize);

        // Set ray origin to sampled lens point
        *origin = camera.position +
                  lensPoint.x * camera.orientation.c0 +
                  lensPoint.y * camera.orientation.c1;

        // Calculate focus point at specified distance
        Point3D focusPoint = camera.position + imagePlanePoint * camera.focusDistance;
        *direction = normalize (focusPoint - *origin);
    }
    else
    {
        // Pinhole camera if lens size is 0
        *origin = camera.position;
        *direction = normalize (imagePlanePoint);
    }
}

// This function is for computing the direct lighting on a surface point (shadigPoint)
// from a light source (lightSample) using a BRDF (bsdf).
// It takes into account visibility, distances, and angles to compute the final light
// contribution at that point. Works for both environment and area lights.
CUDA_DEVICE_FUNCTION CUDA_INLINE RGB computeDirectLighting (
    const Point3D& shadingPoint, const Vector3D& vOutLocal, const ReferenceFrame& shadingFrame,
    const DisneyPrincipled& bsdf, const ripr_shared::LightSample& lightSample)
{
    // Calculate the direction of the shadow ray
    Vector3D shadowRayDir = lightSample.atInfinity ? Vector3D (lightSample.position) : (lightSample.position - shadingPoint);

    // Calculate the distance squared and distance between the light and the shading point
    float dist2 = shadowRayDir.sqLength();
    float dist = std::sqrt (dist2);

    // Normalize the shadow ray direction
    shadowRayDir /= dist;

    // Convert shadow ray direction to local coordinate system
    Vector3D shadowRayDirLocal = shadingFrame.toLocal (shadowRayDir);

    // Compute the cosine of the angle between the light direction and light normal
    float lpCos = dot (-shadowRayDir, lightSample.normal);

    // Compute the cosine of the angle between shadow ray and normal at the shading point in local coords
    float spCos = shadowRayDirLocal.z;

    // Initialize visibility to 1 (completely visible)
    float visibility = 1.0f;

    // Set a high distance for lights at infinity
    if (lightSample.atInfinity)
        dist = 1e+10f;

    // Perform visibility ray tracing to check if the light is occluded
    ripr_shared::VisibilityRayPayloadSignature::trace (
        ripr_plp.f->travHandle,
        shadingPoint.toNative(), shadowRayDir.toNative(), 0.0f, dist * 0.9999f, 0.0f,
        0xFF, OPTIX_RAY_FLAG_NONE,
        RayType::RayType_Visibility, ripr_shared::NumRayTypes, RayType::RayType_Visibility,
        visibility);

    // If the point is visible and faces the light
    if (visibility > 0 && lpCos > 0)
    {
        // Calculate emittance assuming the light is a diffuse emitter
        RGB Le = lightSample.emittance / Pi;

        // Evaluate the  BRDF
        RGB fsValue = bsdf.evaluate (vOutLocal, shadowRayDirLocal);

        // Calculate the geometry term
        float G = lpCos * std::fabs (spCos) / dist2;

        // Final lighting contribution
        RGB ret = fsValue * Le * G;
        return ret;
    }
    else
    {
        // Return black if the point is not visible or does not face the light
        return RGB (0.0f, 0.0f, 0.0f);
    }
}

// This function samples an environmental light based on a set of
// random numbers (u0 and u1) and an importance map.It returns the
// sampled light direction, emittance, and some other attributes
// in lightSample.It also returns the probability density of the
// sampled area in areaPDensity.
CUDA_DEVICE_FUNCTION CUDA_INLINE void sampleEnviroLight (
    const Point3D& shadingPoint,
    float ul, bool sampleEnvLight, float u0, float u1,
    ripr_shared::LightSample* lightSample, float* areaPDensity)
{
    hipTextureObject_t texEmittance = 0;          // Texture object for light emittance
    RGB emittance (0.0f, 0.0f, 0.0f); // Light emittance color
    Point2D texCoord;                      // Texture coordinates

    float u, v;  // Parameters for sampling
    float uvPDF; // PDF for UV sampling

    // Sample the importance map to get UV coordinates and PDF
    ripr_plp.s->envLightImportanceMap.sample (u0, u1, &u, &v, &uvPDF);

    // Convert UV to spherical coordinates
    float phi = 2 * Pi * u;
    float theta = Pi * v;
    if (theta == 0.0f)
    {
        // fix for NAN
        *areaPDensity = 0.0f;
        return;
    }

    // Apply rotation to the environment light
    float posPhi = phi - ripr_plp.f->envLightRotation;
    posPhi = posPhi - floorf (posPhi / (2 * Pi)) * 2 * Pi;

    // Convert spherical to Cartesian coordinates
    Vector3D direction = fromPolarYUp (posPhi, theta);
    Point3D position (direction.x, direction.y, direction.z);

    // Set light sample attributes
    lightSample->position = position;
    lightSample->atInfinity = true;
    lightSample->normal = Normal3D (-position);

    // convert the PDF in texture space to one with respect to area.
    // The true value is: lim_{l to inf} uvPDF / (2 * Pi * Pi * sin(theta)) / l^2
    const float sinTheta = std::sin (theta);
    if (sinTheta == 0.0f)
    {
        *areaPDensity = 0.0f;
        return;
    }

    // Compute the area PDF
    *areaPDensity = uvPDF / (2 * Pi * Pi * std::sin (theta));

    //  printf ("areaPDensity: %f\n", *areaPDensity);

    // Retrieve the environment light texture
    texEmittance = ripr_plp.s->envLightTexture;

    // Set a base emittance value
    emittance = RGB (Pi * ripr_plp.f->envLightPowerCoeff);
    texCoord.x = u;
    texCoord.y = v;

    // If a texture is available, update emittance based on texture values
    if (texEmittance)
    {
        float4 texValue = tex2DLod<float4> (texEmittance, texCoord.x, texCoord.y, 0.0f);
        emittance *= RGB (texValue.x, texValue.y, texValue.z);

        if (isnan (emittance.r) || isnan (emittance.g) || isnan (emittance.b))
        {
            printf ("enviro texture emittance: %f, %f, %f\n", emittance.r, emittance.g, emittance.b);
        }
    }

    // Set the emittance in the light sample
    lightSample->emittance = emittance;
}

// Sample area lights in the scene
CUDA_DEVICE_FUNCTION CUDA_INLINE void sampleAreaLight(
    const Point3D& shadingPoint,
    float ul, float u0, float u1,
    ripr_shared::LightSample* lightSample, float* areaPDensity) 
{
    *areaPDensity = 0.0f;
    
    if (!ripr_plp.s->enableAreaLights || ripr_plp.s->numLightInsts == 0) {
        return;
    }
    
    float lightProb = 1.0f;
    
    // First, sample an instance from the light instance distribution
    float instProb;
    float uGeomInst;
    const uint32_t instSlot = ripr_plp.s->lightInstDist.sample(ul, &instProb, &uGeomInst);
    lightProb *= instProb;
    
    if (instProb == 0.0f) {
        return;
    }
    
    const shared::InstanceData& inst = ripr_plp.s->instanceDataBufferArray[ripr_plp.f->bufferIndex][instSlot];
    
    // Next, sample a geometry instance from this instance
    float geomInstProb;
    float uPrim;
    const uint32_t geomInstIndexInInst = inst.lightGeomInstDist.sample(uGeomInst, &geomInstProb, &uPrim);
    const uint32_t geomInstSlot = inst.geomInstSlots[geomInstIndexInInst];
    lightProb *= geomInstProb;
    
    if (geomInstProb == 0.0f) {
        return;
    }
    
    const shared::GeometryInstanceData& geomInst = ripr_plp.s->geometryInstanceDataBuffer[geomInstSlot];
    
    // Finally, sample a primitive from the geometry instance
    float primProb;
    const uint32_t primIndex = geomInst.emitterPrimDist.sample(uPrim, &primProb);
    lightProb *= primProb;
    
    // Get the triangle and its vertices
    const shared::Triangle& tri = geomInst.triangleBuffer[primIndex];
    const shared::Vertex& vA = geomInst.vertexBuffer[tri.index0];
    const shared::Vertex& vB = geomInst.vertexBuffer[tri.index1];
    const shared::Vertex& vC = geomInst.vertexBuffer[tri.index2];
    
    // Transform vertices to world space
    const Point3D pA = transformPointFromObjectToWorldSpace(vA.position);
    const Point3D pB = transformPointFromObjectToWorldSpace(vB.position);
    const Point3D pC = transformPointFromObjectToWorldSpace(vC.position);
    
    // Sample point on triangle using uniform barycentric sampling
    float sqrtU0 = sqrtf(u0);
    float bc0 = 1.0f - sqrtU0;
    float bc1 = u1 * sqrtU0;
    float bc2 = 1.0f - bc0 - bc1;
    
    // Compute sampled position
    lightSample->position = bc0 * pA + bc1 * pB + bc2 * pC;
    
    // Compute normal (average of vertex normals, transformed to world)
    Normal3D nA = transformNormalFromObjectToWorldSpace(vA.normal);
    Normal3D nB = transformNormalFromObjectToWorldSpace(vB.normal);
    Normal3D nC = transformNormalFromObjectToWorldSpace(vC.normal);
    lightSample->normal = normalize(bc0 * nA + bc1 * nB + bc2 * nC);
    
    // Get material emittance
    const shared::DisneyData& mat = ripr_plp.s->materialDataBuffer[geomInst.materialSlot];
    
    // Sample emittance texture if available
    Point2D texCoord = bc0 * vA.texCoord + bc1 * vB.texCoord + bc2 * vC.texCoord;
    if (mat.emissive) {
        float4 texValue = tex2DLod<float4>(mat.emissive, texCoord.x, texCoord.y, 0.0f);
        lightSample->emittance = RGB(texValue.x, texValue.y, texValue.z) * ripr_plp.s->areaLightPowerCoeff;
    } else {
        lightSample->emittance = RGB(0.0f, 0.0f, 0.0f);
    }
    
    lightSample->atInfinity = false;
    
    // Compute area of the triangle in world space
    Vector3D edge1 = pB - pA;
    Vector3D edge2 = pC - pA;
    float area = 0.5f * length(cross(edge1, edge2));
    
    // Final area PDF
    *areaPDensity = lightProb / area;
}

// Next Event Estimation (NEE) is a technique used in path tracing to improve
// the convergence of the rendered image. Instead of randomly bouncing rays around the scene,
// NEE takes a shortcut and directly samples a light source to check if it contributes to
// the illumination of a point.

// In a traditional path tracer, rays are shot from the camera and bounce around the scene
// until they hit a light source. This can take many bounces and lead to a noisy image.

// With NEE, when a ray hits a surface, the algorithm also sends a direct ray to a light source
// to see if it's visible from that point. This helps to quickly account for direct illumination,
// making the image converge faster and reducing noise.

// This function is for performing Next Event Estimation (NEE) in path tracing.
// It samples a light source, computes the direct lighting from that source,
// and combines it with the BRDF and visibility information.The function also
// uses Multiple Importance Sampling (MIS)
// to balance the contributions from the BRDF and the light source.
CUDA_DEVICE_FUNCTION CUDA_INLINE RGB performNextEventEstimation (
    const Point3D& shadingPoint, const Vector3D& vOutLocal, const ReferenceFrame& shadingFrame,
    const DisneyPrincipled& bsdf,
    PCG32RNG& rng)
{
    RGB ret (0.0f); // Initialize the return value

    // Determine light type sampling probability based on availability
    float envLightProb = 0.0f;
    float areaLightProb = 0.0f;
    
    if (ripr_plp.f->enableEnvLight && ripr_plp.s->envLightTexture) {
        envLightProb = 0.5f;  // Could be based on relative power
    }
    
    if (ripr_plp.s->enableAreaLights && ripr_plp.s->numLightInsts > 0) {
        areaLightProb = 0.5f;
    }
    
    // Normalize probabilities
    float totalProb = envLightProb + areaLightProb;
    if (totalProb == 0.0f) {
        return RGB(0.0f);  // No lights available
    }
    
    envLightProb /= totalProb;
    areaLightProb /= totalProb;
    
    // Select light type
    float uLightType = rng.getFloat0cTo1o();
    bool selectEnvLight = uLightType < envLightProb;
    
    ripr_shared::LightSample lightSample; // Sampled light information
    float areaPDensity = 0.0f;      // Area probability density
    float probToSampleCurLightType = selectEnvLight ? envLightProb : areaLightProb;
    
    if (selectEnvLight && envLightProb > 0.0f) {
        // Sample environment light
        sampleEnviroLight (
            shadingPoint,
            rng.getFloat0cTo1o(), true, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
            &lightSample, &areaPDensity);
    } else if (areaLightProb > 0.0f) {
        // Sample area light
        sampleAreaLight(
            shadingPoint,
            rng.getFloat0cTo1o(), rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
            &lightSample, &areaPDensity);
    }
    
    if (areaPDensity <= 0.0f) {
        return RGB(0.0f);
    }

    areaPDensity *= probToSampleCurLightType; // Update the area PDF with the light type selection probability

    // Calculate the shadow ray direction
    Vector3D shadowRay = lightSample.atInfinity ? Vector3D (lightSample.position) : (lightSample.position - shadingPoint);
    float dist2 = shadowRay.sqLength();                   // Distance squared to the light
    shadowRay /= std::sqrt (dist2);                       // Normalize the shadow ray
    Vector3D vInLocal = shadingFrame.toLocal (shadowRay); // Convert to local coordinates

    float bsdfPDensity = bsdf.evaluatePDF (vOutLocal, vInLocal);
    if (!isfinite (bsdfPDensity) || bsdfPDensity <= 0.0f)
    {
        return RGB (0.0f); // Invalid sampling case, skip contribution
    }

    // Calculate the light source PDF and MIS weight
    float lightPDensity = areaPDensity;
    float misWeight = pow2 (lightPDensity) / (pow2 (bsdfPDensity) + pow2 (lightPDensity));

    // Compute the direct lighting contribution if the area PDF is positive
    if (areaPDensity > 0.0f)
        ret = computeDirectLighting (
                  shadingPoint, vOutLocal, shadingFrame, bsdf, lightSample) *
              (misWeight / areaPDensity);

    return ret; // Return the final lighting contribution
}

// Define the ray generating kernel for path tracing
CUDA_DEVICE_KERNEL void RT_RG_NAME (pathTracing)()
{
    // Get the launch index for this thread
    uint2 launchIndex = make_uint2 (optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    // Initialize the random number generator
    PCG32RNG rng = ripr_plp.s->rngBuffer[launchIndex];

    Point3D origin;
    Vector3D direction;
    const PerspectiveCamera& camera = ripr_plp.f->camera;

    // different approach for DOF
    if (ripr_plp.f->camera.lensSize > 0.0f)
    {
        // Use R2 sequence for pixel sampling with DOF
        uint32_t pixelIndex = launchIndex.y * ripr_plp.s->imageSize.x + launchIndex.x;
        uint32_t sampleIndex = pixelIndex + ripr_plp.f->numAccumFrames * (ripr_plp.s->imageSize.x * ripr_plp.s->imageSize.y);
        float2 r2Sample = R2Sequence(sampleIndex);
        
        Point2D pixel (
            (launchIndex.x + r2Sample.x) / ripr_plp.s->imageSize.x,
            (launchIndex.y + r2Sample.y) / ripr_plp.s->imageSize.y);

        generateCameraRay (rng, ripr_plp.f->camera, pixel, &origin, &direction);
    }
    else
    {
        // Generate jitter offsets using R2 sequence for better distribution
        uint32_t pixelIndex = launchIndex.y * ripr_plp.s->imageSize.x + launchIndex.x;
        uint32_t sampleIndex = pixelIndex + ripr_plp.f->numAccumFrames * (ripr_plp.s->imageSize.x * ripr_plp.s->imageSize.y);
        float2 r2Sample = R2Sequence(sampleIndex);
        
        float jx = r2Sample.x;
        float jy = r2Sample.y;

        // Update the RNG buffer (still needed for other sampling)
        ripr_plp.s->rngBuffer.write (launchIndex, rng);

        // Compute normalized screen coordinates
        float x = (launchIndex.x + jx) / ripr_plp.s->imageSize.x;
        float y = (launchIndex.y + jy) / ripr_plp.s->imageSize.y;

        // Compute vertical and horizontal view angles
        float vh = 2 * std::tan (ripr_plp.f->camera.fovY * 0.5f);
        float vw = ripr_plp.f->camera.aspect * vh;

        // Setup ray origin and direction
        origin = camera.position;
        direction = normalize (camera.orientation * Vector3D (vw * (0.5f - x), vh * (0.5f - y), 1));
    }
    
    //// Debug: Print traversable handle and ray info for first pixel (similar to Shocker)
    //if (launchIndex.x == 0 && launchIndex.y == 0) {
    //    printf("RiPREngine RG: travHandle=%llu, origin=(%.2f,%.2f,%.2f), dir=(%.2f,%.2f,%.2f)\n",
    //           ripr_plp.f->travHandle, origin.x, origin.y, origin.z, direction.x, direction.y, direction.z);
    //}

    // Initialize ray payload
    SearchRayPayload payload;
    payload.alpha = RGB (1.0f, 1.0f, 1.0f);
    payload.contribution = RGB (0.0f, 0.0f, 0.0f);
    payload.pathLength = 1;
    payload.prevDirPDensity = 1.0f;  // Camera rays have uniform PDF
    payload.deltaSampled = 0;
    payload.terminate = false;
    SearchRayPayload* payloadPtr = &payload;

    RGB firstHitAlbedo (0.0f, 0.0f, 0.0f);
    Normal3D firstHitNormal (0.0f, 0.0f, 0.0f);
    RGB* firstHitAlbedoPtr = &firstHitAlbedo;
    Normal3D* firstHitNormalPtr = &firstHitNormal;

    // Initialize variables for storing hit point properties
    HitPointParams hitPointParams;
    hitPointParams.positionInWorld = Point3D (NAN);
    hitPointParams.prevPositionInWorld = Point3D (NAN);
    hitPointParams.normalInWorld = Normal3D (NAN);
    hitPointParams.texCoord = Point2D (NAN);
    HitPointParams* hitPointParamsPtr = &hitPointParams;

    // Main path tracing loop
    while (true)
    {
        // Trace the ray and collect results
        SearchRayPayloadSignature::trace (
            ripr_plp.f->travHandle, origin.toNative(), direction.toNative(),
            0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
            RayType_Search, NumRayTypes, RayType_Search,
            rng, payloadPtr, hitPointParamsPtr, firstHitAlbedoPtr, firstHitNormalPtr);

        // Break out of the loop if conditions are met
        if (payload.terminate || payload.pathLength >= ripr_plp.f->maxPathLength)
            break;

        // Update ray origin and direction for the next iteration
        origin = payload.origin;
        direction = payload.direction;
        ++payload.pathLength;
    }

    // Store the updated RNG state back to the buffer
    ripr_plp.s->rngBuffer[launchIndex] = rng;

    RGB prevAlbedoResult (0.0f, 0.0f, 0.0f);
    RGB prevColorResult (0.0f, 0.0f, 0.0f);
    Normal3D prevNormalResult (0.0f, 0.0f, 0.0f);

    if (ripr_plp.f->numAccumFrames > 0)
    {
        prevColorResult = RGB (getXYZ (ripr_plp.s->beautyAccumBuffer.read (launchIndex)));
        prevAlbedoResult = RGB (getXYZ (ripr_plp.s->albedoAccumBuffer.read (launchIndex)));
        prevNormalResult = Normal3D (getXYZ (ripr_plp.s->normalAccumBuffer.read (launchIndex)));
    }

    float curWeight = 1.0f / (1 + ripr_plp.f->numAccumFrames);

    // Clamp contribution to reduce fireflies
    RGB clampedContribution = payload.contribution;
    clampedContribution.r = fminf(clampedContribution.r, ripr_plp.s->maxRadiance);
    clampedContribution.g = fminf(clampedContribution.g, ripr_plp.s->maxRadiance);
    clampedContribution.b = fminf(clampedContribution.b, ripr_plp.s->maxRadiance);

    RGB colorResult = (1 - curWeight) * prevColorResult + curWeight * clampedContribution;
#if 0
    if (isnan (colorResult.r) || isnan (colorResult.g) || isnan (colorResult.b))
    {
        // Add this line to print the payload.contribution values
        printf ("payload.contribution: %f, %f, %f\n", payload.contribution.r, payload.contribution.g, payload.contribution.b);
        colorResult = RGB (make_float3 (1000000.0f, 0.0f, 0.0f)); // super red
    }
    else if (isinf (colorResult.r) || isinf (colorResult.g) || isinf (colorResult.b))
    {
        printf ("payload.contribution: %f, %f, %f\n", payload.contribution.r, payload.contribution.g, payload.contribution.b);
        colorResult = RGB (make_float3 (0.0f, 1000000.0f, 0.0f)); // super green
    }
    else if (colorResult.r < 0.0f || colorResult.g < 0.0f || colorResult.b < 0.0f)
    {
        printf ("payload.contribution is negative: %f, %f, %f\n", payload.contribution.r, payload.contribution.g, payload.contribution.b);
        colorResult = RGB (make_float3 (0.0f, 0.0f, 1000000.0f)); // super blue
    }
#endif
    RGB albedoResult = (1 - curWeight) * prevAlbedoResult + curWeight * firstHitAlbedo;

#if 0
    if (albedoResult.r < 0.0f || albedoResult.r > 1.0f
        || albedoResult.g < 0.0f || albedoResult.g > 1.0f 
        || albedoResult.b < 0.0f || albedoResult.b > 1.0f)
        {
            // Add this line to print the payload.contribution values
            printf ("firstHitAlbedo  %f, %f, %f\n", firstHitAlbedo.r, firstHitAlbedo.g, firstHitAlbedo.b);
            albedoResult = RGB (make_float3 (1000000.0f, 0.0f, 0.0f)); // super red
        }

#endif

    Normal3D normalResult = (1 - curWeight) * prevNormalResult + curWeight * firstHitNormal;
#if 0
    if (isnan (normalResult.x) || isnan (normalResult.y) || isnan (normalResult.z))
    {
        // Add this line to print the payload.contribution values
        printf ("firstHitNormal: %f, %f, %f\n", firstHitNormal.x, firstHitNormal.y, firstHitNormal.z);
            normalResult = Normal3D (make_float3 (1000000.0f, 0.0f, 0.0f)); // super red
    }
#endif
    ripr_plp.s->beautyAccumBuffer.write (launchIndex, make_float4 (colorResult.toNative(), 1.0f));
    ripr_plp.s->albedoAccumBuffer.write (launchIndex, make_float4 (albedoResult.toNative(), 1.0f));
    ripr_plp.s->normalAccumBuffer.write (launchIndex, make_float4 (normalResult.toNative(), 1.0f));
    
    // Calculate motion vectors
    Vector2D motionVector (0.0f, 0.0f);
    if (!isnan (hitPointParams.positionInWorld.x) && !isnan (hitPointParams.prevPositionInWorld.x))
    {
        // Current pixel position (center of pixel)
        Point2D curRasterPos (launchIndex.x + 0.5f, launchIndex.y + 0.5f);
        
        // Calculate previous frame position using previous camera
        Point2D prevRasterPos = ripr_plp.f->prevCamera.calcScreenPosition (hitPointParams.prevPositionInWorld) 
                               * Point2D (ripr_plp.s->imageSize.x, ripr_plp.s->imageSize.y);
        
        // Motion vector is the difference
        motionVector = curRasterPos - prevRasterPos;
    }
    
    // Write motion vector to flow accumulation buffer
    ripr_plp.s->flowAccumBuffer.write (launchIndex, make_float4 (motionVector.x, motionVector.y, 0.0f, 1.0f));
}
// Miss shader that handles environment lighting and background
CUDA_DEVICE_KERNEL void RT_MS_NAME (miss)()
{
    // Get payload data
    SearchRayPayload* payload;
    HitPointParams* hitPntParams;
    SearchRayPayloadSignature::get (nullptr, &payload, &hitPntParams, nullptr, nullptr);

    // Store normalized direction as surface normal
    Vector3D vOut (-Vector3D (optixGetWorldRayDirection()));
    hitPntParams->normalInWorld = Normal3D (vOut);

    // Calculate raw HDR environment value without power coefficient
    RGB environmentValue (0.0f, 0.0f, 0.0f);
    float theta = 0.0f;
    Point2D texCoord (0.0f, 0.0f);

    if (ripr_plp.s->envLightTexture)
    {
        Vector3D rayDir = normalize (Vector3D (optixGetWorldRayDirection()));
        float posPhi;
        toPolarYUp (rayDir, &posPhi, &theta);
        float phi = posPhi + ripr_plp.f->envLightRotation;
        phi = phi - floorf (phi / (2 * Pi)) * 2 * Pi;
        // Clamp texture coordinates to [0, 1) range to prevent evaluatePDF assertion
        float u = fminf(phi / (2 * Pi), 0.999999f);
        float v = fminf(theta / Pi, 0.999999f);
        texCoord = Point2D (u, v);
        float4 texValue = tex2DLod<float4> (ripr_plp.s->envLightTexture, texCoord.x, texCoord.y, 0.0f);
        environmentValue = RGB (texValue.x, texValue.y, texValue.z);
    }

    // For background color, use raw HDR or solid color without power coefficient
    RGB background;
    if (ripr_plp.s->useSolidBackground || !ripr_plp.s->envLightTexture)
    {
        background = RGB (ripr_plp.s->backgroundColor.x, ripr_plp.s->backgroundColor.y, ripr_plp.s->backgroundColor.z);
    }
    else
    {
        background = environmentValue; // Use raw environment value for background
    }

    // Apply MIS weight and power coefficient for surface lighting only
    float misWeight = 1.0f;
    if (payload->pathLength > 1 && !payload->deltaSampled)
    {
        float uvPDF = ripr_plp.s->envLightImportanceMap.evaluatePDF (texCoord.x, texCoord.y);
        float hypAreaPDensity = uvPDF / (2 * Pi * Pi * std::sin (theta));
        float lightPDensity = hypAreaPDensity;
        if (ripr_plp.s->lightInstDist.integral() > 0.0f)
        {
            lightPDensity *= probToSampleEnvLight;
        }
        float bsdfPDensity = 0.25f;
        misWeight = pow2 (bsdfPDensity) / (pow2 (bsdfPDensity) + pow2 (lightPDensity));

        // Apply power coefficient only for surface lighting
        payload->contribution += payload->alpha * (environmentValue * ripr_plp.f->envLightPowerCoeff) * misWeight;
    }
    else
    {
        // First bounce - use raw background without power coefficient
       payload->contribution = background;
       // payload->contribution = RGB(1.0f, 0.0f, 0.0f) * ripr_plp.envLightPowerCoeff;
    }

    payload->terminate = true;
}

CUDA_DEVICE_KERNEL void RT_CH_NAME (shading)()
{
    // Get material and geometry instance data from global buffers
    auto sbtr = HitGroupSBTRecordData::get();
    const shared::DisneyData& mat = ripr_plp.s->materialDataBuffer[sbtr.materialSlot];
    const shared::GeometryInstanceData& geomInst = ripr_plp.s->geometryInstanceDataBuffer[sbtr.geomInstSlot];
    
    // Get instance data using buffer index from launch parameters
    const uint32_t bufIdx = ripr_plp.f->bufferIndex;  
    const shared::InstanceData& inst = ripr_plp.s->instanceDataBufferArray[bufIdx][optixGetInstanceId()];

    // Initialize random number generator and payload
    PCG32RNG rng;
    SearchRayPayload* payload;
    RGB* firstHitAlbedo;
    Normal3D* firstHitNormal;
    HitPointParams* hitPntParams;
    SearchRayPayloadSignature::get (&rng, &payload, &hitPntParams, &firstHitAlbedo, &firstHitNormal);

    // Calculate hit point parameters
    auto hp = HitPointParameter::get();
    Point3D positionInWorld;
    Normal3D shadingNormalInWorld;
    Vector3D texCoord0DirInWorld;
    Normal3D geometricNormalInWorld;
    Point2D texCoord;
    float hypAreaPDensity;
    computeSurfacePoint (
        geomInst, hp.primIndex, hp.bcB, hp.bcC,
        Point3D (optixGetWorldRayOrigin()),
        &positionInWorld, &shadingNormalInWorld, &texCoord0DirInWorld,
        &geometricNormalInWorld, &texCoord, &hypAreaPDensity);

    // Setup shading frame
    Vector3D vOut = normalize (-Vector3D (optixGetWorldRayDirection()));
    float frontHit = dot (vOut, geometricNormalInWorld) >= 0.0f ? 1.0f : -1.0f;
    ReferenceFrame shadingFrame (shadingNormalInWorld, texCoord0DirInWorld);

    // Offset hit point to avoid self-intersection
    positionInWorld = offsetRayOrigin (positionInWorld, frontHit * geometricNormalInWorld);
    Vector3D vOutLocal = shadingFrame.toLocal (vOut);
    
    // Debug for center pixel - same as Shocker
    bool debugPixel = (optixGetLaunchIndex().x == 512 && optixGetLaunchIndex().y == 384);
    if (debugPixel && payload->pathLength == 1) {
        printf("[RiPR] Geometry: vOut=(%.3f,%.3f,%.3f), vOutLocal=(%.3f,%.3f,%.3f), normal=(%.3f,%.3f,%.3f)\n",
            vOut.x, vOut.y, vOut.z,
            vOutLocal.x, vOutLocal.y, vOutLocal.z,
            shadingNormalInWorld.x, shadingNormalInWorld.y, shadingNormalInWorld.z);
        printf("[RiPR] TexCoord0Dir=(%.3f,%.3f,%.3f)\n",
            texCoord0DirInWorld.x, texCoord0DirInWorld.y, texCoord0DirInWorld.z);
    }
    
    // Calculate previous position for motion vectors (only on first hit)
    if (payload->pathLength == 1)
    {
        hitPntParams->positionInWorld = positionInWorld;
        hitPntParams->prevPositionInWorld = inst.curToPrevTransform * positionInWorld;
        hitPntParams->normalInWorld = shadingNormalInWorld;
        hitPntParams->texCoord = texCoord;
    }

    // Create DisneyPrincipled instance directly instead of using BSDF
    DisneyPrincipled bsdf = DisneyPrincipled::create (
        mat, texCoord, 0.0f, ripr_plp.s->makeAllGlass, ripr_plp.s->globalGlassIOR,
        ripr_plp.s->globalTransmittanceDist, ripr_plp.s->globalGlassType);

    // Delta in PBR rendering refers to a perfect specular reflection or transmission that occurs at a single angle.
    // It represents an infinitely narrow spike of reflection, like what you'd see in a perfect mirror, where all light
    // reflects at exactly the angle predicted by the law of reflection. In rendering systems, delta distributions are
    // handled as special cases since they can't be sampled like regular BRDFs. They're primarily used to model
    // idealized surfaces like perfect mirrors, smooth glass, and pristine metals.
    bool isDeltaMaterial = mat.transparency > 0.9f && mat.metallic <= 0.0f;

    // Handle emissive surfaces
    RGB emission = bsdf.evaluateEmission();
    if (emission.r > 0.0f || emission.g > 0.0f || emission.b > 0.0f)
    {
        if (payload->pathLength == 1)
        {
            // Direct camera hit - no MIS needed
            payload->contribution += payload->alpha * emission;
        }
        else if (ripr_plp.s->enableAreaLights && !payload->deltaSampled)
        {
            // Indirect hit with MIS
            // We need to compute the probability of having sampled this light via NEE
            
            // First check if this instance is emissive
            if (inst.isEmissive)
            {
                // Compute area of this triangle
                const shared::Triangle& tri = geomInst.triangleBuffer[hp.primIndex];
                const shared::Vertex& v0 = geomInst.vertexBuffer[tri.index0];
                const shared::Vertex& v1 = geomInst.vertexBuffer[tri.index1];
                const shared::Vertex& v2 = geomInst.vertexBuffer[tri.index2];
                
                Point3D p0 = transformPointFromObjectToWorldSpace(v0.position);
                Point3D p1 = transformPointFromObjectToWorldSpace(v1.position);
                Point3D p2 = transformPointFromObjectToWorldSpace(v2.position);
                
                float area = 0.5f * length(cross(p1 - p0, p2 - p0));
                
                // Get the various sampling probabilities
                float instProb = ripr_plp.s->lightInstDist.evaluatePMF(optixGetInstanceId());
                float geomInstProb = inst.lightGeomInstDist.evaluatePMF(0); // Assuming single geom inst
                float primProb = geomInst.emitterPrimDist.evaluatePMF(hp.primIndex);
                
                // Light sampling PDF (area measure)
                float lightPDF = instProb * geomInstProb * primProb / area;
                
                // Account for light type selection probability
                float envLightProb = (ripr_plp.f->enableEnvLight && ripr_plp.s->envLightTexture) ? 0.5f : 0.0f;
                float areaLightProb = 1.0f - envLightProb;
                lightPDF *= areaLightProb;
                
                // Convert to solid angle measure
                Point3D prevPos = payload->origin - payload->direction * 0.001f; // Approximate previous position
                Vector3D toLight = positionInWorld - prevPos;
                float dist2 = toLight.sqLength();
                float cosTheta = std::abs(dot(normalize(toLight), geometricNormalInWorld));
                lightPDF *= dist2 / std::max(cosTheta, 1e-6f);
                
                // BSDF PDF from previous direction
                float bsdfPDF = payload->prevDirPDensity;
                
                // MIS weight (power heuristic)
                float misWeight = bsdfPDF * bsdfPDF / (bsdfPDF * bsdfPDF + lightPDF * lightPDF);
                
                payload->contribution += payload->alpha * emission * misWeight;
            }
            else
            {
                // Non-emissive instance hit but material is emissive - use full contribution
                payload->contribution += payload->alpha * emission;
            }
        }
        else
        {
            // Delta sampled or area lights disabled - no MIS
            payload->contribution += payload->alpha * emission;
        }
    }
    // Only do NEE for non-delta materials
    if (!isDeltaMaterial)
    {
        payload->contribution += payload->alpha * performNextEventEstimation (positionInWorld, vOutLocal, shadingFrame, bsdf, rng);
    }

    // Sample new direction
    Vector3D vInLocal;
    float dirPDensity;
    RGB sampledValue = bsdf.sampleThroughput (
        vOutLocal, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
        &vInLocal, &dirPDensity);
    
    // Debug for center pixel - same as Shocker
    if (debugPixel) {
        printf("[RiPR] sampleThroughput: throughput=(%.3f,%.3f,%.3f), dirPDF=%.6f, pathLen=%d\n",
            sampledValue.r, sampledValue.g, sampledValue.b, dirPDensity, payload->pathLength);
    }

    if (dirPDensity > 0.0f)
    {
        // Update payload for next bounce
        payload->alpha = payload->alpha * (sampledValue * std::fabs (vInLocal.z) / dirPDensity);
        
        if (debugPixel) {
            RGB newAlpha = payload->alpha;
            printf("[RiPR] After update: alpha=(%.3f,%.3f,%.3f)\n",
                newAlpha.r, newAlpha.g, newAlpha.b);
        }
        
        payload->origin = positionInWorld + shadingNormalInWorld * (vInLocal.z > 0 ? 0.001f : -0.001f);
        payload->direction = shadingFrame.fromLocal (vInLocal);
        payload->prevDirPDensity = dirPDensity;
        payload->deltaSampled = isDeltaMaterial;
        payload->terminate = false;

        // Store first hit data
        if (payload->pathLength == 1)
        {
            if (isDeltaMaterial)
            {
                // For perfectly transparent materials, use base color
                *firstHitAlbedo = RGB (mat.baseColor);
            }
            else
            {
                *firstHitAlbedo = bsdf.evaluateDHReflectanceEstimate (vOutLocal);
            }
            *firstHitNormal = shadingNormalInWorld;
        }
    }
    else
    {
        payload->terminate = true;
    }

    SearchRayPayloadSignature::set (&rng, nullptr, nullptr, nullptr, nullptr);
}


// Determines how light passes through transparent objects for more accurate shadows
CUDA_DEVICE_KERNEL void RT_AH_NAME (visibility)()
{
    // Get material and geometry instance data
    auto sbtr = HitGroupSBTRecordData::get();
    const shared::DisneyData& mat = ripr_plp.s->materialDataBuffer[sbtr.materialSlot];
    const shared::GeometryInstanceData& geomInst = ripr_plp.s->geometryInstanceDataBuffer[sbtr.geomInstSlot];

    // Get barycentric coordinates
    float2 bc = optixGetTriangleBarycentrics();

    // Get UV coordinates from hit point
    auto hp = HitPointParameter::get();
    const Triangle& tri = geomInst.triangleBuffer[hp.primIndex];
    const Vertex& v0 = geomInst.vertexBuffer[tri.index0];
    const Vertex& v1 = geomInst.vertexBuffer[tri.index1];
    const Vertex& v2 = geomInst.vertexBuffer[tri.index2];

    float b0 = 1.0f - (bc.x + bc.y);
    Point2D texCoord = b0 * v0.texCoord + bc.x * v1.texCoord + bc.y * v2.texCoord;

    // Read material properties at this point
    float transparency = tex2DLod<float> (mat.transparency, texCoord.x, texCoord.y, 0.0f);
    if (mat.useAlphaForTransparency)
    {
        float4 baseColorValue = tex2DLod<float4> (mat.baseColor, texCoord.x, texCoord.y, 0.0f);
        float alpha = baseColorValue.w;

        // For binary alpha, use threshold approach
        float alphaThreshold = 0.5f;
        if (alpha < alphaThreshold)
        {
            // Make fully transparent
            transparency = 1.0f;
        }
        else if (transparency < 0.1f)
        {
            // Only override if not already transparent
            transparency = 0.0f;
        }
    }
    float transmittance = tex2DLod<float> (mat.transmittance, texCoord.x, texCoord.y, 0.0f);
    float transmittanceDistance = tex2DLod<float> (mat.transmittanceDistance, texCoord.x, texCoord.y, 0.0f);
    transmittanceDistance = 0.5f;
    float4 baseColorValue = tex2DLod<float4> (mat.baseColor, texCoord.x, texCoord.y, 0.0f);
    RGB baseColor (baseColorValue.x, baseColorValue.y, baseColorValue.z);

    // Get current visibility value
    float visibility = 0.0f;
    VisibilityRayPayloadSignature::get (&visibility);

  
    // Skip if the material is opaque (no transparency)
    if (transparency <= 0.0f)
    {
        visibility = 0.0f;
        VisibilityRayPayloadSignature::set (&visibility);
        optixTerminateRay();
        return;
    }

    // Calculate ray direction and normal for Fresnel calculations
    Vector3D rayDir = normalize (Vector3D (optixGetWorldRayDirection()));
    Normal3D normal = normalize (b0 * v0.normal + bc.x * v1.normal + bc.y * v2.normal);
    normal = normalize (transformNormalFromObjectToWorldSpace (normal));

    // Ensure normal faces against ray direction
    float NdotI = dot (normal, rayDir);
    if (NdotI > 0.0f)
        normal = -normal;

    // Calculate Fresnel for incident ray
    float cosTheta = abs (dot (normal, rayDir));
    float ior = ripr_plp.s->globalGlassIOR; // Use global glass IOR from pipeline params
    float F = mx_fresnel_dielectric (cosTheta, ior);

    // Calculate how much light passes through (transmission)
    float transmission = (1.0f - F) * transparency * transmittance;

    // Apply color absorption using Beer's law if not thin-walled
    bool thinWalled = (ripr_plp.s->globalGlassType == 0);
    if (!thinWalled && transmittanceDistance > 0.0f)
    {
        // Estimate approximate ray distance through the material
        // This is a simplification; for accurate results we'd need entry/exit points
        float estDistance = transmittanceDistance;

        // Apply Beer's law: T = exp(-absorption * distance)
        RGB transmissionColor;
        transmissionColor.r = exp (-baseColor.r * estDistance);
        transmissionColor.g = exp (-baseColor.g * estDistance);
        transmissionColor.b = exp (-baseColor.b * estDistance);

        // Convert RGB transmission to scalar (using luminance formula)
        float coloredTransmission = 0.2126f * transmissionColor.r +
                                    0.7152f * transmissionColor.g +
                                    0.0722f * transmissionColor.b;

        // Apply colored absorption to transmission
        transmission *= coloredTransmission;
    }

    // Update visibility based on transmission
    visibility *= transmission;
    VisibilityRayPayloadSignature::set (&visibility);

    // Continue ray if we still have meaningful visibility
    if (visibility > 0.01f)
        return;

    // Terminate ray if visibility too low
    visibility = 0.0f;
    VisibilityRayPayloadSignature::set (&visibility);
    optixTerminateRay();
}


#if 0
// FIXME this needs work.
CUDA_DEVICE_KERNEL void RT_AH_NAME (visibility)()
{
    // Get material and geometry instance data
    auto sbtr = HitGroupSBTRecordData::get();
    const shared::DisneyData& mat = ripr_plp.s->materialDataBuffer[sbtr.materialSlot];
    const shared::GeometryInstanceData& geomInst = ripr_plp.s->geometryInstanceDataBuffer[sbtr.geomInstSlot];

    // Get barycentric coordinates
    float2 bc = optixGetTriangleBarycentrics();

    // Get UV coordinates from hit point
    auto hp = HitPointParameter::get();
    const Triangle& tri = geomInst.triangleBuffer[hp.primIndex];
    const Vertex& v0 = geomInst.vertexBuffer[tri.index0];
    const Vertex& v1 = geomInst.vertexBuffer[tri.index1];
    const Vertex& v2 = geomInst.vertexBuffer[tri.index2];

    float b0 = 1.0f - (bc.x + bc.y);
    Point2D texCoord = b0 * v0.texCoord + bc.x * v1.texCoord + bc.y * v2.texCoord;

    // Read transparency value
    float transparency = tex2DLod<float> (mat.transparency, texCoord.x, texCoord.y, 0.0f);

    // Get current visibility value
    float visibility = 0.0f;
    VisibilityRayPayloadSignature::get (&visibility);

    if (transparency > 0.0f)
    {
        // Attenuate visibility by transparency
        visibility *= transparency;
        VisibilityRayPayloadSignature::set (&visibility);

        // Continue ray if still enough visibility
        if (visibility > 0.01f)
            return;
    }

    // Terminate ray if opaque or visibility too low
    visibility = 0.0f;
    VisibilityRayPayloadSignature::set (&visibility);
    optixTerminateRay();
}

#endif